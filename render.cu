#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>

__global__ void getIterationCounts(double x0, double y0, double xD, double yD, int nCols, int nRows, int limitIter, int* iterations)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x >= nCols || y >= nRows) {return;}; // banish bad threads

	double zxOld = 0;
	double zyOld = 0;
	double zxNew = 0;
	double zyNew = 0;

	double cx = x0 + ((double) x) * xD;
	double cy = y0 - ((double) y) * yD;

	int escIter = -1; // Will vary from 0 (immediately outside the escape radius) to maxIter - 1; unless it never escapes, then it is -1

	for (int i = 0; i < limitIter; i++) {
		zxNew = ((zxOld * zxOld) - (zyOld * zyOld) + cx);
		zyNew = ((2 * zxOld * zyOld) + cy);
		if ((zxNew * zxNew + zyNew * zyNew) > 4) {escIter = i; break;}; // escape radius is 2. Therefore the square of the escape is 4
		zxOld = zxNew;
		zyOld = zyNew;
	}

	iterations[x + y * nCols] = escIter;
}

template<class T>
__global__ void getBlockwiseExtrema(const T* inputArray, T* blockwiseExtrema, int inputLength, int numBlocks, bool min)
{
    extern __shared__ float blockInput[];

    int globalIdx = threadIdx.x + blockIdx.x * blockDim.x;
    int localIdx  = threadIdx.x;
    int numBlockThreads = (blockIdx.x != numBlocks - 1) ? blockDim.x : (inputLength - ((numBlocks - 1) * blockDim.x));

    if (globalIdx >= inputLength) return; // banish bad threads

    blockInput[localIdx] = inputArray[globalIdx];
    __syncthreads();

    T curr;
    T neww;

    for (int i = 1; i < numBlockThreads; i *= 2) {
        if (localIdx > (i - 1)) {
            curr = blockInput[localIdx - i];
            neww = blockInput[localIdx];
            if (curr < 0) curr = 0;
            if (neww < 0) neww = 0;
            if (min) blockInput[localIdx] = (neww < curr) ? neww : curr;
            else blockInput[localIdx] = (neww > curr) ? neww : curr;
            __syncthreads();
        }
    }

    int last = numBlockThreads - 1;
    if (localIdx == last) blockwiseExtrema[blockIdx.x] = blockInput[last];
}

__global__ void colorImage(unsigned char* image, int* iterations, int nCols, int inputSize, double minIter, double maxIter, double minHue, double maxHue)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int globalIdx = x + y * nCols;
	if (globalIdx >= inputSize) {return;}; // banish bad threads
	if (iterations[globalIdx] < 0) {return;}; // banish threads for areas that should be black

	double H = minHue + ((iterations[globalIdx] - minIter) / (maxIter - minIter)) * (maxHue - minHue);
	double S = 1;
	double V = 1;

	while (H < 0) {H += 360; };
	while (H >= 360) {H -= 360; };
	double R, G, B;
	if (V <= 0) {
		R = 0;
		G = 0;
		B = 0;
	}
	else if (S <= 0) {
		R = G = B = V;
	}
	else
	{
		double hf = H / 60.0;
		int i = (int) floor(hf);
		double f = hf - i;
		double pv = V * (1 - S);
		double qv = V * (1 - S * f);
		double tv = V * (1 - S * (1 - f));

		switch (i) {
	      	 // Red is the dominant color
	    	case 0:
	    		R = V;
	      		G = tv;
	      		B = pv;
	        break;
	        // Green is the dominant color
	    	case 1:
	    		R = qv;
	    		G = V;
	    		B = pv;
	    		break;
	    	case 2:
	    		R = pv;
	    		G = V;
	    		B = tv;
	    		break;
	    	// Blue is the dominant color
	    	case 3:
	    		R = pv;
	    		G = qv;
	    		B = V;
	    		break;
	    	case 4:
	    		R = tv;
	    		G = pv;
	    		B = V;
	    		break;
	    	// Red is the dominant color
	    	case 5:
	    		R = V;
	    		G = pv;
	    		B = qv;
	    		break;
	    	// Just in case we overshoot on our math by a little, we put these here. Since its a switch it won't slow us down at all to put these here.
	    	case 6:
	    		R = V;
	    		G = tv;
	    		B = pv;
	    		break;
	    	case -1:
	    		R = V;
	    		G = pv;
	    		B = qv;
	    		break;
	    	// The color is not defined, we should throw an error.
	    	default:
	    	  //LFATAL("i Value error in Pixel conversion, Value is %d", i);
	    	  R = G = B = V; // Just pretend its black/white
	    	  break;
		}
	}
	unsigned char r = (unsigned char) (R * 255.0);
	unsigned char g = (unsigned char) (G * 255.0);
	unsigned char b = (unsigned char) (B * 255.0);

	if (r > 255) r = 255;
	if (g > 255) g = 255;
	if (b > 255) b = 255;

	image[3 * globalIdx    ] = b;
	image[3 * globalIdx + 1] = g;
	image[3 * globalIdx + 2] = r;
}

void render(unsigned char* h_image, double x0, double y0, double xD, double yD, int nCols, int nRows, int limitIter)
{
	int pointsPerBlock = 32;

	// Determine kernel properties
	dim3 threadDims(pointsPerBlock, pointsPerBlock);
	int xBlocks = nCols / pointsPerBlock;
	int yBlocks = nRows / pointsPerBlock;
	if (nCols % pointsPerBlock != 0) xBlocks++;
	if (nRows % pointsPerBlock != 0) yBlocks++;
	dim3 blockDims(xBlocks, yBlocks);

	int* d_iterations;
	hipMalloc(&d_iterations, nCols * nRows * sizeof(int));
	getIterationCounts<<<blockDims, threadDims>>>(x0, y0, xD, yD, nCols, nRows, limitIter, d_iterations);
	hipDeviceSynchronize();
	printf("Calculated iteration counts\n");

	int nThreads = threadDims.x * threadDims.y;
	int nBlocks = (nCols * nRows) / nThreads;
	if (nThreads % (nCols * nRows) != 0) nBlocks++;
	int* blockwiseExtrema;
	hipMalloc(&blockwiseExtrema, nBlocks * sizeof(int));

	/*
	int* d_min; // Error in oversizing the grid size
	int h_min[1];
	hipMalloc(&d_min, sizeof(int));
	getBlockwiseExtrema<int><<<nBlocks, nThreads, nThreads * sizeof(int)>>>(d_iterations, blockwiseExtrema, nCols * nRows, nBlocks, true);
	hipDeviceSynchronize();
	getBlockwiseExtrema<int><<<1, nBlocks, nBlocks * sizeof(int)>>>(blockwiseExtrema, d_min, nBlocks, 1, true);
	hipDeviceSynchronize();
	hipMemcpy(h_min, d_min, sizeof(int), hipMemcpyDeviceToHost);

	int* d_max;
	int h_max[1];
	hipMalloc(&d_max, sizeof(int));
	getBlockwiseExtrema<int><<<nBlocks, nThreads, nThreads * sizeof(int)>>>(d_iterations, blockwiseExtrema, nCols * nRows, nBlocks, false);
	hipDeviceSynchronize();
	getBlockwiseExtrema<int><<<1, nBlocks, nBlocks * sizeof(int)>>>(blockwiseExtrema, d_max, nBlocks, 1, false);
	hipDeviceSynchronize();
	hipMemcpy(h_max, d_max, sizeof(int), hipMemcpyDeviceToHost);
	printf("Calculated iteration range\n");

	double minIter = (double) h_min[0];
	double maxIter = (double) h_max[0];
	*/

	// manual ovverride of min / max
	double minIter = 0;
	double maxIter = limitIter;

	unsigned char* d_image;
	hipMalloc(&d_image, 3 * nCols * nRows * sizeof(unsigned char));
	hipMemset(d_image, 0, 3 * nCols * nRows * sizeof(unsigned char));
	printf("Allocated memory for image\n");
	colorImage<<<blockDims, threadDims>>>(d_image, d_iterations, nCols, nCols * nRows, minIter, maxIter, 0.0, 200.0);
	hipDeviceSynchronize();
	printf("Colored image\n");
	hipMemcpy(h_image, d_image, 3 * nCols * nRows * sizeof(unsigned char), hipMemcpyDeviceToHost);
	printf("Moved image to host\n");

    /* DEBUG iteration count
	int h_iterations[nCols * nRows];
	hipMemcpy(h_iterations, d_iterations, nCols * nRows * sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < nRows; i++) {
		for (int j = 0; j < nCols; j++) {
			printf("%3d", h_iterations[j + i * nCols]);
		}
		printf("\n");
	}
	printf("\n\n");
	*/

	/* DEBUG min/max
	printf("Min: %d\n", h_min[0]);
	printf("Max: %d\n\n\n", h_max[0]);
	*/

	/* DEBUG image gneration
	for (int i = 0; i < nRows; i++) {
		for (int j = 0; j < nCols; j++) {
			printf("[");
			for (int k = 0; k < 3; k++) {
				printf("%3u ", h_image[3 * (j + i * nCols) + k]);
			}
			printf("] ");
		}
		printf("\n");
	}
	printf("\n\n");
	*/
}
